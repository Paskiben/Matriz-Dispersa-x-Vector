#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#define BSIZE 1024
#define PRINT 0

// Funcion para la multiplicacion en CPU
void mulMdCPU(const float *CSR, const float *V,const float *CI, const float *RI, float *answer, const long int n);
// Kernel, multipicacion en GPU
__global__ void mulMdGPU(const float *CSR, const float *V, const float *CI, const float *RI, float *answer, const long int n);
// Funciones para imprimir la Matriz
void printMatrix(float *M, long int n);
// Funcion para imprimir el vector
void printVector(float *V, long n);

int main(int argc, char** argv) {
    
    // Comprobacion de los argumentos
    if(argc < 6){
        fputs_unlocked("Debe ejecutarse como ./prog <n> <d> <m> <s> <nt>\n", stdout);
        exit(EXIT_FAILURE);
    }
    
    // Recibir argumentos
    long int n = atoi(argv[1]);
    float d = atof(argv[2]);
    int m = atoi(argv[3]);
    float s = atof(argv[4]);
    int nt = atoi(argv[5]);
    omp_set_num_threads(nt);
    srand(s); // Semilla

    // Md = matriz dispersa con valores nulos (contiene la data de la matriz), V = vector a calcular, RI = Row Index, answer = respuesta.
    float *Md = new float[n*n]{0}, *V = new float[n],
    *RI = new float[n+1], *answer = new float[n]{},
    *dV, *dRI, *danswer;
    long int nelem=0;

    // Iniciar la distribucion random de la matriz y el vector (CPU)
    printf("inicializando...."); fflush(stdout);
    double t1 = omp_get_wtime();
    for(int i=0; i<n; ++i){
        V[i] = rand();
        for(int j=0; j<n; ++j){
            if((float)rand()/RAND_MAX<=d){
                Md[i*n + j] = rand();
                nelem++;
            }
        }
    }

    // Imprimir la matriz y el vector
    if (PRINT){
        fputs_unlocked("\n", stdout);
        printMatrix(Md, n);
        fputs_unlocked("\n", stdout);
        printVector(V, n);
    }
    
    // CSR  = Vector de los valores no nulos de la matriz, CI = Column Index
    float *CSR = new float[nelem], *CI = new float[nelem],
     *dCSR, *dCI;
    int k = 0;
    
    // Alocacion de datos del CSR y CI
    for(int i = 0; i < n; ++i){
        RI[i] = k;
        for(int j = 0; j < n; ++j)
            if(Md[i*n + j] != 0){
                CSR[k] = Md[i*n + j];
                CI[k++] = j; 
            }
    }
    RI[n] = k;
    delete[] Md;

    double t2 = omp_get_wtime();
    printf("ok: %f secs\n", t2-t1); fflush(stdout);

    // Alocar memoria en device  (GPU)
    hipMalloc(&dCSR, sizeof(float) * nelem);
    hipMalloc(&dV, sizeof(float) * n);
    hipMalloc(&dCI, sizeof(float) * nelem);
    hipMalloc(&dRI, sizeof(float) * (n+1));
    hipMalloc(&danswer, sizeof(float) * n);

    // Copiar de Host -> Device
    hipMemcpy(dCSR, CSR, sizeof(float)*nelem, hipMemcpyHostToDevice);
    hipMemcpy(dV, V, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(dCI, CI, sizeof(float)*nelem, hipMemcpyHostToDevice);
    hipMemcpy(dRI, RI, sizeof(float)*(n+1), hipMemcpyHostToDevice);

    // Definition of block and grid sizes
    dim3 block(BSIZE, 1, 1);  // (x, y, z) --> bloque de x * y * z threads
    dim3 grid((n + BSIZE-1)/BSIZE, 1, 1);
    
    // Se crea un evento en cuda
    hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	printf("calculando...."); fflush(stdout);
	hipEventRecord(start);

    // Modo GPU o CPU
	if (m) {
		printf("GPU\n"); fflush(stdout);
		mulMdGPU<<<grid, block>>>(dCSR, dV, dCI, dRI, danswer, n);
	}
	else {
		printf("CPU\n"); fflush(stdout);
		mulMdCPU(CSR, V, CI, RI, answer, n);
	}
    // Inicialisacion mediciones de tiempo
	hipDeviceSynchronize(); hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
    
    // Recuperar los datos del device a la RAM
	if (m) { hipMemcpy(answer, danswer, sizeof(float)*n, hipMemcpyDeviceToHost); }
    
    // Calculo de tiempos de ejecucion
	float time = milliseconds/1000.0f;
	float tflops = ((float)n*n*(2*n)/time)/(1e12);
	printf("ok: %f secs (%f TFLOPS)\n", time, tflops); fflush(stdout);
    if (PRINT) {fputs_unlocked("\n", stdout); printVector(answer, n);}
    
    // Borrar los arreglos dinamicos
    delete[] CSR; delete[] V; delete[] CI; delete[] RI; delete[] answer;
}

// Funcion para el calculo paralelo de CPU de la matriz 
void mulMdCPU(const float *CSR, const float *V,const float *CI, const float *RI, float *answer, const long int n) {
    #pragma omp parallel for
    for(int i=0; i <= n; ++i){
        float tempAnswer=0.0f;
        for(int j=RI[i]; j<RI[i+1]; ++j)
            tempAnswer += CSR[j]*V[(int)CI[j]];
        answer[i] = tempAnswer;
    }
}

// Funcion para el calculo paralelo de GPU de la matriz 
__global__ void mulMdGPU(const float *CSR, const float *V, const float *CI, const float *RI, float *answer, const long int n) {
    int tidx = (blockDim.x * blockIdx.x)  + threadIdx.x;
    if (tidx < (n)) {
        float tempAnswer=0.0f;
        for (int i = RI[tidx]; i < RI[tidx + 1]; ++i)
            tempAnswer += CSR[i] * V[(int)CI[i]];
        answer[tidx] = tempAnswer;
    }
}

// Funcion para imprir Matrices
void printMatrix(float *M, long int n) {
    for(int i = 0; i < n; ++i){
        for(int j = 0; j<n; ++j)
            printf("%.0f ", M[i*n + j]);
        fputs_unlocked("\n", stdout);
    }
}

// Funcion para imprir Vectores
void printVector(float *V, long n) {
    for(int i=0; i<n; ++i)
        printf("%.0f ", V[i]);
    fputs_unlocked("\n", stdout);
}