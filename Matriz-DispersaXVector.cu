#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#define BSIZE 1024
#define PRINT 0
using namespace std;

void mulMdCPU(float *&CSR, float *&V, float *&CI, float *&RI, float *&answer, long n);
__global__ void mulMdGPU(float *CSR, float *V, float *CI, float *RI, float *answer, long int n);

void printMatrix(float *&M, long n){
    for(int i = 0; i < n; i++){
        for(int j=0; j<n; ++j)
            printf("%d ", M[i*n + j]);
        fputs_unlocked("\n", stdout);
    }
}

void printVector(float *&M, long n){
    for(int i=0; i<n; ++i)
        printf("%d ", M[i]);
    fputs_unlocked("\n", stdout);
}

int main(int argc, char** argv) {
    if(argc <6){
        fputs_unlocked("Debe ejecutarse como ./prog <n> <d> <m> <s> <nt>", stdout);
        exit(EXIT_FAILURE);
    }

    long int n = atoi(argv[1]);
    float d = atof(argv[2]);
    int m = atoi(argv[3]);
    float s = atof(argv[4]);
    int nt = atoi(argv[5]);
    omp_set_num_threads(nt);
    srand(s);

    //md = matriz dispersa (contiene la data de la matriz), v = vector a calcular, CI = Column Index, RI = Row Index, R = respuesta.
    float *Md = new float[n*n]{}, *V = new float[n],
    *RI = new float[n+1], *answer = new float[n]{},
    *dV, *dRI, *danswer;
    long int nelem=0;
    
    // inicializar arreglos en Host (CPU)
    double t1 = omp_get_wtime();
    //Inicialisacion de las matrices
    printf("inicializando...."); fflush(stdout);
    fputs_unlocked("\n", stdout);
    int x, y;
    //#pragma omp parallel for
    for(int i=0; i<n; ++i){
        x = rand()%10;
        V[i] = x; 
        for(int j=0; j<n; ++j){
            if((float)rand()/RAND_MAX<=d){
                y = rand()%10 + 1;
                Md[i*n + j] = y;
                nelem++;
            }
        }
    }
    if(PRINT){
        printMatrix(Md, n);
        fputs_unlocked("\n", stdout);
        printVector(V, n);
    }
    
    float *CSR = new float[nelem], *CI = new float[nelem],
    *dCSR, *dCI;
    int k = 0;
    //#pragma omp parallel for
    for(int i = 0; i < n; i++){
        RI[i]= k;
        for(int j=0; j<n; ++j){
            if(Md[i*n + j] != 0){
                CSR[k] = Md[i*n + j];
                CI[k] = j; 
                k++;
            }
        }
    }
    RI[n]= k;
    delete(Md);

    double t2 = omp_get_wtime();
    printf("ok: %f secs\n", t2-t1); fflush(stdout);

    // allocar memoria en device  (GPU)
    hipMalloc(&dCSR, sizeof(float) * nelem);
    hipMalloc(&dV, sizeof(float) * n);
    hipMalloc(&dCI, sizeof(float) * nelem);
    hipMalloc(&dRI, sizeof(float) * (n+1));
    hipMalloc(&danswer, sizeof(float) * n);

    // copiar de Host -> Device
    hipMemcpy(dCSR, CSR, sizeof(float)*nelem, hipMemcpyHostToDevice);
    hipMemcpy(dV, V, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(dCI, CI, sizeof(float)*nelem, hipMemcpyHostToDevice);
    hipMemcpy(dRI, RI, sizeof(float)*(n+1), hipMemcpyHostToDevice);

    //Definition of block and grid sizes
    dim3 block(BSIZE, 1, 1);  // (x, y, z) --> bloque de x * y * z threads
    dim3 grid((n + BSIZE-1)/BSIZE, 1, 1);

    hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	printf("calculando...."); fflush(stdout);
	hipEventRecord(start);
	if(m){
		printf("GPU\n"); fflush(stdout);
		mulMdGPU<<<grid, block>>>(dCSR, dV, dCI, dRI, danswer, n);
	}
	else{
		printf("CPU\n"); fflush(stdout);
		mulMdCPU(CSR, V, CI, RI, answer, n);
	}
	hipDeviceSynchronize(); hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	if(m){ hipMemcpy(answer, danswer, sizeof(float)*n, hipMemcpyDeviceToHost); }
	float time = milliseconds/1000.0f;
	float tflops = ((float)n*n*(2*n)/time)/(1e12);
	printf("ok: %f secs (%f TFLOPS)\n", time, tflops); fflush(stdout);
    if(PRINT){fputs_unlocked("\n", stdout); printVector(answer, n);}
    
    delete(CSR); delete(V); delete(CI); delete(RI); delete(answer);
}

void mulMdCPU(float *&CSR, float *&V, float *&CI, float *&RI, float *&answer, long int n) {
    #pragma omp parallel for
    for(int i=0;i<n+1;++i){
        for(int j=RI[i];j<RI[i+1];++j)
            answer[i] += CSR[j]*V[(int)CI[j]];
    }
}

__global__ void mulMdGPU(float *CSR, float *V, float *CI, float *RI, float *answer, long int n) {
    int tidx = (blockDim.x * blockIdx.x)  + threadIdx.x;
    if (tidx < (n)) {
        for (int i = RI[tidx]; i < RI[tidx + 1]; ++i) {
            answer[tidx] += CSR[i] * V[(int)CI[i]];
        }
    }
}