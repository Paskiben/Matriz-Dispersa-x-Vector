#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#define BSIZE 16
using namespace std;


int main(int argc, char** argv) {
    if(argc <6){
        fputs_unlocked("Debe ejecutarse como ./prog <n> <d> <m> <s> <nt>", stdout);
        exit(EXIT_FAILURE);
    }
    long n = atoi(argv[1]);
    float d = atoi(argv[2]);
    int m = atoi(argv[3]);
    float s = atoi(argv[4]);
    int nt = atoi(argv[5]);
    omp_set_num_threads(nt);
    srand(s);
    long nelem = n*n;

    //Se cran las matrices a multiplicar y en la que se guardara el resultado
    float *md, *V, *CI, *RI, *R, rowCont=0;
    Md = new float[nelem]; V = new float[nelem];
    CI = new float[nelem]; RI = new float[n]; R = new float[n];

    // inicializar arreglos en Host (CPU)
    double t1 = omp_get_wtime();
    //Inicialisacion de las matrices
    printf("inicializando...."); fflush(stdout);
    #pragma omp parallel for
    for(int i=0; i<n; ++i){
        RI[i]=rowCont;
        R[i]=0;
        V[i] = rand();
        for(int j=0; j<n; ++j){
            if(rand()/RAND_MAX<d)
            else{
                Md[i*n + j] = rand();
                CI[i*n + j] = i
                ++rowCont;
            }
        }
    }
    

    double t2 = omp_get_wtime();
    printf("ok: %f secs\n", t2-t1); fflush(stdout);

    // allocar memoria en device  (GPU)
    hipMalloc(&dMd, sizeof(float) * nelem);
    hipMalloc(&dV, sizeof(float) * n);
    hipMalloc(&dCI, sizeof(float) * nelem);
    hipMalloc(&dRI, sizeof(float) * n);
    hipMalloc(&dR, sizeof(float) * n);

    // copiar de Host -> Device
    hipMemcpy(dMd, Md, sizeof(float)*nelem, hipMemcpyHostToDevice);
    hipMemcpy(dV, V, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(dCI, CI, sizeof(float)*nelem, hipMemcpyHostToDevice);
    hipMemcpy(dRI, RI, sizeof(float)*n, hipMemcpyHostToDevice);


    dim3 block(BSIZE, BSIZE, 1);  // (x, y, z) --> bloque de x * y * z threads
    dim3 grid((n + BSIZE-1)/BSIZE, (n + BSIZE-1)/BSIZE, 1);

    

    hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	printf("calculando...."); fflush(stdout);
	hipEventRecord(start);
	if(m){
		printf("GPU\n"); fflush(stdout);
		//mikernel<<<grid, block>>>(dA, dB, dC, n);
	}
	else{
		printf("CPU\n"); fflush(stdout);
		//cpu(A, B, C, n);	
	}
	hipDeviceSynchronize(); hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	if(m){ hipMemcpy(C, dC, sizeof(float)*nelem, hipMemcpyDeviceToHost); }
	float time = milliseconds/1000.0f;
	float tflops = ((float)n*n*(2*n)/time)/(1e12);
	printf("ok: %f secs (%f TFLOPS)\n", time, tflops); fflush(stdout);
	print_mat(C, n, "MATRIX C");
}

void multiplicarMatrizDispersaCPU(float& *md, float& *V, float& *CI, float& *RI, float& *R,long n){
    #pragma omp parallel for 
    for(int i=0;i<n;++i){
        s
    }
}

void
      push_back(const value_type& __x)
      {
	if (this->_M_impl._M_finish != this->_M_impl._M_end_of_storage)
	  {
	    _GLIBCXX_ASAN_ANNOTATE_GROW(1);
	    _Alloc_traits::construct(this->_M_impl, this->_M_impl._M_finish,
				     __x);
	    ++this->_M_impl._M_finish;
	    _GLIBCXX_ASAN_ANNOTATE_GREW(1);
	  }
	else
	  _M_realloc_insert(end(), __x);
      }