#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#define BSIZE 16
using namespace std;


int main(int argc, char** argv) {
    if(argc <6){
        fputs_unlocked("Debe ejecutarse como ./prog <n> <d> <m> <s> <nt>", stdout);
        exit(EXIT_FAILURE);
    }
    long n = atoi(argv[1]);
    float d = atoi(argv[2]);
    int m = atoi(argv[3]);
    float s = atoi(argv[4]);
    int nt = atoi(argv[5]);
    omp_set_num_threads(nt);
    srand(s);
    long nelem = n*n;

    //md = matriz dispersa (contiene la data de la matriz), v = vector a calcular, CI = Column Index, RI = Row Index, R = respuesta.
    float *Md, *V, *CI, *RI, *R, rowCont=0;
    Md = new float[nelem]; V = new float[nelem];
    CI = new float[nelem]; RI = new float[n]; R = new float[n];

    // inicializar arreglos en Host (CPU)
    double t1 = omp_get_wtime();
    //Inicialisacion de las matrices
    printf("inicializando...."); fflush(stdout);
    #pragma omp parallel for
    for(int i=0; i<n; ++i){
        //RI[i]= rowCont;
        R[i]= 0;
        V[i] = rand();
        for(int j=0; j<n; ++j){
            if(rand()/RAND_MAX>d)
                Md[i*n + j]=0;
            else{
                Md[i*n + j] = rand();
                //CI[i*n + j] = i;
                ++rowCont;
            }
        }
    }
    float *CSR = 0;
    CSR = new float[rowCont];
    CI = new float[rowCont];
    int k = 0;
    #pragma omp parallel for collapse(2)
    for(int i = 0; i < n; i++){
        for(int j=0; j<n; ++j){
            if(Md[i*n + j] != 0){
                CSR[k] = Md[i*n + j];
                CI[k] = i*n + j;
                k++;
            }
        }
    }
    double t2 = omp_get_wtime();
    printf("ok: %f secs\n", t2-t1); fflush(stdout);

    // allocar memoria en device  (GPU)
    hipMalloc(&dMd, sizeof(float) * nelem);
    hipMalloc(&dV, sizeof(float) * n);
    hipMalloc(&dCI, sizeof(float) * nelem);
    hipMalloc(&dRI, sizeof(float) * n);
    hipMalloc(&dR, sizeof(float) * n);

    // copiar de Host -> Device
    hipMemcpy(dMd, Md, sizeof(float)*nelem, hipMemcpyHostToDevice);
    hipMemcpy(dV, V, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(dCI, CI, sizeof(float)*nelem, hipMemcpyHostToDevice);
    hipMemcpy(dRI, RI, sizeof(float)*n, hipMemcpyHostToDevice);


    dim3 block(BSIZE, BSIZE, 1);  // (x, y, z) --> bloque de x * y * z threads
    dim3 grid((n + BSIZE-1)/BSIZE, (n + BSIZE-1)/BSIZE, 1);

    

    hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	printf("calculando...."); fflush(stdout);
	hipEventRecord(start);
	if(m){
		printf("GPU\n"); fflush(stdout);
		//mikernel<<<grid, block>>>(dA, dB, dC, n);
	}
	else{
		printf("CPU\n"); fflush(stdout);
		//cpu(A, B, C, n);	
	}
	hipDeviceSynchronize(); hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	if(m){ hipMemcpy(C, dC, sizeof(float)*nelem, hipMemcpyDeviceToHost); }
	float time = milliseconds/1000.0f;
	float tflops = ((float)n*n*(2*n)/time)/(1e12);
	printf("ok: %f secs (%f TFLOPS)\n", time, tflops); fflush(stdout);
	print_mat(C, n, "MATRIX C");
}